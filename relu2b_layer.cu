#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/relu2b_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ReLU2BForward(const int n, const Dtype* in, Dtype* out, Dtype th, Dtype d) {
  CUDA_KERNEL_LOOP(i, n) {
    if (in[i] <= th) {
      out[i] = Dtype(0);
    } else if (in[i] <= 1.5*d) {
      out[i] = Dtype(d);
    } else if (in[i] <= 2.5*d) {
      out[i] = Dtype(2*d);
    } else {
      out[i] = Dtype(3*d);
    }
    //out[index] = in[index] > 0 ? in[index] : in[index] * negative_slope;
  }
}

template <typename Dtype>
void ReLU2BLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  ReLUParameter relu_param = this->layer_param().relu_param();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const Dtype th = relu_param.thresh();
  const Dtype d = relu_param.delta();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ReLU2BForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, th, d);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void ReLU2BBackward(const int n, const Dtype* in_data, const Dtype* in_diff, Dtype* out_diff, const Dtype d) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ((in_data[index] > 0) * (in_data[index] < 3*d));
  }
}

template <typename Dtype>
void ReLU2BLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  ReLUParameter relu_param = this->layer_param().relu_param();
  const Dtype d = relu_param.delta();
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    ReLU2BBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, top_diff, bottom_diff, d);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(ReLU2BLayer);


}  // namespace caffe
